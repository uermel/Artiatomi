#include "hip/hip_runtime.h"
//  Copyright (c) 2018, Michael Kunz and Frangakis Lab, BMLS,
//  Goethe University, Frankfurt am Main.
//  All rights reserved.
//  http://kunzmi.github.io/Artiatomi
//  
//  This file is part of the Artiatomi package.
//  
//  Artiatomi is free software: you can redistribute it and/or modify
//  it under the terms of the GNU General Public License as published by
//  the Free Software Foundation, either version 3 of the License, or
//  (at your option) any later version.
//  
//  Artiatomi is distributed in the hope that it will be useful,
//  but WITHOUT ANY WARRANTY; without even the implied warranty of
//  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//  GNU General Public License for more details.
//  
//  You should have received a copy of the GNU General Public License
//  along with Artiatomi. If not, see <http://www.gnu.org/licenses/>.
//  
////////////////////////////////////////////////////////////////////////


#ifndef WBPWEIGHTING_CU
#define WBPWEIGHTING_CU

//Includes for IntelliSense 
#include "hip/hip_runtime.h"
#include ""
#include <>
#include <surface_functions.h>


#include <stdio.h>
#include "hipfft/hipfft.h"

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif


// transform vector by matrix

enum FilterMethod
{
	FM_RAMP,
	FM_EXACT,
	FM_CONTRAST2,
	FM_CONTRAST10,
	FM_CONTRAST30
};

__device__ float sinc(float x)
{
	float res = 1;
	if (x != 0)
	{
		res = sinf(M_PI * x) / (M_PI * x);
	}
	return res;
}

extern "C"
__global__ 
void wbpWeighting(hipComplex* img, size_t stride, unsigned int pixelcount, float psiAngle, FilterMethod fm, int proj_index, int projectionCount, float thickness, const float* __restrict__ tiltAngles)
{
	//compute x,y,z indiced
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;	
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (x >= pixelcount/2 + 1) return;
	if (y >= pixelcount) return;

	float xpos = x;
	float ypos = y;
	if (ypos > pixelcount * 0.5f)
		ypos = (pixelcount - ypos) * -1.0f;

	float temp = xpos;
	float sinus =  __sinf(psiAngle);
	float cosin =  __cosf(psiAngle);

	xpos = cosin * xpos - sinus * ypos;
	ypos = sinus * temp + cosin * ypos;

	float length = ypos / (pixelcount / 2.0f);
	float weight = 1;
	switch (fm)
	{
	case FM_RAMP:
		weight = fminf(abs(length), 1.0f);
		break;
	case FM_EXACT:
		{
			//psiAngle += M_PI * 0.5f;
			float x_st = -ypos * cos(tiltAngles[proj_index])*sin(psiAngle) - xpos *cos(psiAngle);
			float y_st =  ypos * cos(tiltAngles[proj_index])*cos(psiAngle) - xpos *sin(psiAngle);
			float z_st =  ypos * sin(tiltAngles[proj_index]);

            //float x_st = -xpos * cos(tiltAngles[proj_index])*sin(psiAngle) - ypos *cos(psiAngle);
            //float y_st =  xpos * cos(tiltAngles[proj_index])*cos(psiAngle) - ypos *sin(psiAngle);
            //float z_st =  xpos * sin(tiltAngles[proj_index]);

			float w = 0;

			for (int tilt = 0; tilt < projectionCount; tilt++)
			{
				if (tilt != proj_index && tiltAngles[tilt] != -999.0f)
				{
					// Berechnung der geometrischen Distanz zu der Ebene
					float d_tmp = x_st*sin(tiltAngles[tilt])*sin(psiAngle) - y_st*sin(tiltAngles[tilt])*cos(psiAngle) + z_st*cos(tiltAngles[tilt]);

					float d2 = abs(sin(tiltAngles[tilt])) * thickness + cos(tiltAngles[tilt]);

					if (abs(d_tmp) > d2)
						d_tmp = d2;

					w += sinc(d_tmp / d2);
				}
			}
			// Normalize, such that the center is 1 / number of projections, and
			// the boundary is one!!
			w += 1.0f;
			w = 1.0f / w;

			//Added normalization(zero frequencies set to zero)
			if (ypos == 0)
			{
				w = 0;
			}
			weight = w;
		}
		break;
	case FM_CONTRAST2:
		{//1.000528623371163   0.006455924123082   0.005311341463650   0.001511856638478 1024
		 //1.000654227857550   0.006008581017124   0.004159659493151   0.000975903396538 1856
			const float p1 = 1.000654227857550f;
			const float p2 = 0.006008581017124f;
			const float p3 = 0.004159659493151f;
			const float p4 = 0.000975903396538f;
			if (length == 0)
			{
				weight = 0;
			}
			else
			{
				float logfl = logf(abs(length));
				weight = p1 + p2 * logfl + p3 * logfl * logfl + p4 * logfl * logfl * logfl;
			}
			weight = fmaxf(0, fminf(weight, 1));
		}
		break;
	case FM_CONTRAST10:		
		{//1.001771328635575   0.019634409648661   0.014871972759515   0.004962873817517 1024
		 //1.003784816598589   0.029016377161629   0.019582940715148   0.004559409669984 1856
			const float p1 = 1.003784816598589f;
			const float p2 = 0.029016377161629f;
			const float p3 = 0.019582940715148f;
			const float p4 = 0.004559409669984f;
			if (length == 0)
			{
				weight = 0;
			}
			else
			{
				float logfl = logf(abs(length));
				weight = p1 + p2 * logfl + p3 * logfl * logfl + p4 * logfl * logfl * logfl;
			}
			weight = fmaxf(0, fminf(weight, 1));
		}
		break;
	case FM_CONTRAST30:		
		{//0.998187224092783   0.019542575617926   0.010359773048706   0.006975890938967 1024
		 //0.999884616010943   0.000307646262566   0.004742915272196   0.004806551368900 1856
			const float p1 = 0.999884616010943f;
			const float p2 = 0.000307646262566f;
			const float p3 = 0.004742915272196f;
			const float p4 = 0.004806551368900f;
			if (length == 0)
			{
				weight = 0;
			}
			else
			{
				float logfl = logf(abs(length));
				weight = p1 + p2 * logfl + p3 * logfl * logfl + p4 * logfl * logfl * logfl;
			}
			weight = fmaxf(0, fminf(weight, 1));
		}
		break;
	}
	
	hipComplex res = *(((hipComplex*)((char*)img + stride * y)) + x);
	res.x *= weight;
	res.y *= weight;
	/*res.x = weight;
	res.y = 0;*/

	*(((hipComplex*)((char*)img + stride * y)) + x) = res;
	
}

extern "C"
__global__
void fourierFilter(float2* img, size_t stride, int pixelcount, float lp, float hp, float lps, float hps)
{
	//compute x,y indices 
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= pixelcount / 2 + 1) return;
	if (y >= pixelcount) return;

	float mx = (float)x;
	float my = (float)y;
	if (my > pixelcount * 0.5f)
		my = (pixelcount - my) * -1.0f;

	float dist = sqrtf(mx * mx + my * my);
	float fil = 0;

	lp = lp - lps;
	hp = hp + hps;
	//Low pass
	if (lp > 0)
	{
		if (dist <= lp) fil = 1;
	}
	else
	{
		if (dist <= pixelcount / 2 - 1) fil = 1;
	}
	//Gauss
	if (lps > 0)
	{
		float fil2;
		if (dist < lp) fil2 = 1;
		else fil2 = 0;

		fil2 = (-fil + 1.0f) * (float)expf(-((dist - lp) * (dist - lp) / (2 * lps * lps)));
		if (fil2 > 0.001f)
			fil = fil2;
	}

	if (lps > 0 && lp == 0 && hp == 0 && hps == 0)
		fil = (float)expf(-((dist - lp) * (dist - lp) / (2 * lps * lps)));

	if (hp > 0)
	{
		float fil2 = 0;
		if (dist >= hp) fil2 = 1;

		fil *= fil2;

		if (hps > 0)
		{
			float fil3 = 0;
			if (dist < hp) fil3 = 1;
			fil3 = (-fil2 + 1) * (float)expf(-((dist - hp) * (dist - hp) / (2 * hps * hps)));
			if (fil3 > 0.001f)
				fil = fil3;
		}
	}

	float2 erg = *(((float2*)((char*)img + stride * y)) + x);
	erg.x *= fil;
	erg.y *= fil;
	*(((float2*)((char*)img + stride * y)) + x) = erg;
}

extern "C"
__global__
void doseWeighting(float2* img, size_t stride, int pixelcount, float dose, float pixelsize)
{
	//compute x,y indices 
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= pixelcount / 2 + 1) return;
	if (y >= pixelcount) return;

	float mx = (float)x;
	float my = (float)y;
	if (my > pixelcount * 0.5f)
		my = (pixelcount - my) * -1.0f;

	float dist = sqrtf(mx * mx + my * my);
	float fil = 0;

	dist = dist / (pixelcount / 2 / pixelsize);
	fil = expf(-dose * dist);

	float2 erg = *(((float2*)((char*)img + stride * y)) + x);
	erg.x *= fil;
	erg.y *= fil;
	*(((float2*)((char*)img + stride * y)) + x) = erg;
}

extern "C"
__global__
void conjMul(float2* complxA, float2* complxB, size_t stride, int pixelcount)
{
	//compute x,y,z indiced 
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= pixelcount / 2 + 1) return;
	if (y >= pixelcount) return;

	float2 a = *(((float2*)((char*)complxA + stride * y)) + x);
	float2 b = *(((float2*)((char*)complxB + stride * y)) + x);
	float2 erg;
	//conj. complex of a: -a.y
	erg.x = a.x * b.x + a.y * b.y;
	erg.y = a.x * b.y - a.y * b.x;
	*(((float2*)((char*)complxA + stride * y)) + x) = erg;

}

extern "C"
__global__
void conjMulPC(float2* complxA, float2* complxB, size_t stride, int pixelcount)
{
    //compute x,y,z indiced
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= pixelcount / 2 + 1) return;
    if (y >= pixelcount) return;

    float2 a = *(((float2*)((char*)complxA + stride * y)) + x);
    float2 b = *(((float2*)((char*)complxB + stride * y)) + x);
    float2 erg;

    //conj. complex of a: -a.y
    erg.x = a.x * b.x + a.y * b.y;
    erg.y = a.x * b.y - a.y * b.x;

    float amplitude = sqrtf(erg.x * erg.x + erg.y * erg.y);

    if (amplitude != 0)
    {
        erg.x /= amplitude;
        erg.y /= amplitude;
    }
    else
    {
        erg.x = erg.y = 0;
    }

    *(((float2*)((char*)complxA + stride * y)) + x) = erg;

}



extern "C"
__global__
void maxShift(float* img, size_t stride, int pixelcount, int maxShift)
{
	//compute x,y,z indiced 
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= pixelcount) return;
	if (y >= pixelcount) return;

	float dist = 0;
	float mx = x;
	float my = y;

	if (mx > pixelcount / 2)
		mx = pixelcount - mx;
	
	if (my > pixelcount / 2)
		my = pixelcount - my;

	dist = sqrtf(mx * mx + my * my);

	if (dist > maxShift)
	{
		*(((float*)((char*)img + stride * y)) + x) = 0;
	}
}

extern "C"
__global__
void maxShiftWeighted(float* img, size_t stride, int pixelcount, int maxShift)
{
	//compute x,y,z indiced 
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= pixelcount) return;
	if (y >= pixelcount) return;

	float dist = 0;
	float mx = x;
	float my = y;

	if (mx > pixelcount / 2)
		mx = pixelcount - mx;

	if (my > pixelcount / 2)
		my = pixelcount - my;

	dist = sqrtf(mx * mx + my * my);

	if (dist > maxShift)
	{
		*(((float*)((char*)img + stride * y)) + x) = 0;
	}
	else
	{
		*(((float*)((char*)img + stride * y)) + x) /= dist+0.0001f;
	}
}

extern "C"
__global__
void findPeak(float* img, size_t stride, char* maskInv, size_t strideMask, int pixelcount, float maxThreshold)
{
	//compute x,y,z indiced 
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= pixelcount) return;
	if (y >= pixelcount) return;

	
	int xm = x - 1;
	int ym = y - 1;
	int xp = x + 1;
	int yp = y + 1;

	//note: the cc map is fft-shifted!
	//wrap negative indices
	if (xm < 0)
	{
		xm = pixelcount - 1;
	}
	if (ym < 0)
	{
		ym = pixelcount - 1;
	}
	if (xp >= pixelcount)
	{
		xp = 0;
	}
	if (yp >= pixelcount)
	{
		yp = 0;
	}
	
	float* rowImgP = (float*)((char*)img + stride * yp);
	float* rowImg = (float*)((char*)img + stride * y);
	float* rowImgM = (float*)((char*)img + stride * ym);
	unsigned char* rowMask = (unsigned char*)((char*)maskInv + strideMask * y);

	float val = rowImg[x];
	if (rowImg[xm] < val && rowImg[xp] < val &&
		rowImgM[x] < val && rowImgP[x] < val &&
		rowImgM[xm] <= val && rowImgP[xm] < val &&
		rowImgM[xp] <= val && rowImgP[xp] < val &&
		val >= maxThreshold
		)
	{
		rowMask[x] = 0;
	}
	else
	{
		rowMask[x] = 1;
	}
}


texture<float, 3, hipReadModeElementType> texVol;
extern "C"
__global__ void rot3d(int size, float3 rotMat0, float3 rotMat1, float3 rotMat2, float* outVol)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

	float center = size / 2;

	float3 vox = make_float3(x - center, y - center, z - center);
	float3 rotVox;
	rotVox.x = center + rotMat0.x * vox.x + rotMat1.x * vox.y + rotMat2.x * vox.z;
	rotVox.y = center + rotMat0.y * vox.x + rotMat1.y * vox.y + rotMat2.y * vox.z;
	rotVox.z = center + rotMat0.z * vox.x + rotMat1.z * vox.y + rotMat2.z * vox.z;

	outVol[z * size * size + y * size + x] = tex3D(texVol, rotVox.x + 0.5f, rotVox.y + 0.5f, rotVox.z + 0.5f);
}

extern "C"
__global__ void sphericalMask3D(int size, float radius, float* outVol)
{
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

    float center = size / 2;

    float3 vox = make_float3(x - center, y - center, z - center);

    float rad = sqrtf(vox.x*vox.x + vox.y*vox.y + vox.z*vox.z);
    if (rad > radius) {
        outVol[z * size * size + y * size + x] = 1;
    }
    else {
        outVol[z * size * size + y * size + x] = 0;
    }
}
#endif


/// Apply mask to a volume split on multiple GPUs
// volume -- CUDA surface object bound to CUDA arrays of the volume on different nodes
// mask -- CUDA device pointer to mask volume (needs to be replicated on each node)
// tempStore -- CUDA device pointer for storing the modified region to revert the volume later
// volmin -- The min global coordinate on this MPI node
// volmax -- The max global coodinate on this MPI node
// dimMask -- Dimensions of the mask volume
// radiusMask -- radius of the mask applied
// centerInVol -- The center of the mask in global volume coordinates (Matlab convention, i.e. 1-based, so -1)
extern "C"
__global__
void applyMask(hipSurfaceObject_t volume, const float* mask, float* tempStore, int3 volmin, int3 volmax, int3 dimMask, int3 radiusMask, int3 centerInVol)
{
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

    // Return if outside mask
    if ((x >= dimMask.x) || (y >= dimMask.y) || (z >= dimMask.z)) return;

    // Compute mask position in complete volume (-1 because Matlab --> C++)
    int3 maskGrid;
    maskGrid.x = x - radiusMask.x + centerInVol.x - 1;
    maskGrid.y = y - radiusMask.y + centerInVol.y - 1;
    maskGrid.z = z - radiusMask.z + centerInVol.z - 1;

    // Return if outside current subvolume
    bool mincond = (maskGrid.x < volmin.x) || (maskGrid.y < volmin.y) || (maskGrid.z < volmin.z);
    bool maxcond = (maskGrid.x >= volmax.x) || (maskGrid.y >= volmax.y) || (maskGrid.z >= volmax.z);
    if (mincond || maxcond) return;

    // Adjust coordinates to current subvolume
    int3 volGrid;
    volGrid.x = maskGrid.x - volmin.x;
    volGrid.y = maskGrid.y - volmin.y;
    volGrid.z = maskGrid.z - volmin.z;

    // Do the deed.
    size_t mask_idx = z * dimMask.x * dimMask.y + y * dimMask.x + x;

    float data;
    surf3Dread<float>(&data, volume, volGrid.x * sizeof(float), volGrid.y, volGrid.z, hipBoundaryModeTrap);
    tempStore[mask_idx] = data;
    data *= mask[mask_idx];
    surf3Dwrite<float>(data, volume, volGrid.x * sizeof(float), volGrid.y, volGrid.z, hipBoundaryModeTrap);
}


extern "C"
__global__
void restoreVolume(hipSurfaceObject_t volume, const float* tempStore, int3 volmin, int3 volmax, int3 dimMask, int3 radiusMask, int3 centerInVol)
{
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

    // Return if outside mask
    if ((x >= dimMask.x) || (y >= dimMask.y) || (z >= dimMask.z)) return;

    // Compute mask position in complete volume (-1 because Matlab --> C++)
    int3 maskGrid;
    maskGrid.x = x - radiusMask.x + centerInVol.x - 1;
    maskGrid.y = y - radiusMask.y + centerInVol.y - 1;
    maskGrid.z = z - radiusMask.z + centerInVol.z - 1;

    // Return if outside current MPI-subvolume
    bool mincond = (maskGrid.x < volmin.x) || (maskGrid.y < volmin.y) || (maskGrid.z < volmin.z);
    bool maxcond = (maskGrid.x >= volmax.x) || (maskGrid.y >= volmax.y) || (maskGrid.z >= volmax.z);
    if (mincond || maxcond) return;

    // Adjust coordinates to current MPI-subvolume
    int3 volGrid;
    volGrid.x = maskGrid.x - volmin.x;
    volGrid.y = maskGrid.y - volmin.y;
    volGrid.z = maskGrid.z - volmin.z;

    // Do the deed.
    size_t mask_idx = z * dimMask.x * dimMask.y + y * dimMask.x + x;

    float data = tempStore[mask_idx];
    surf3Dwrite<float>(data, volume, volGrid.x * sizeof(float), volGrid.y, volGrid.z, hipBoundaryModeTrap);
}