#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif


#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>


extern "C"
__global__ void subKernel(const float* __restrict__ value, float* dataInOut, int size)
{
	const int voxel = blockIdx.x * blockDim.x + threadIdx.x;
	if (voxel >= size)
		return;

	dataInOut[voxel] -= *value;
}

extern "C"
__global__ void subdivKernel(const float* __restrict__ value, float* dataInOut, int size, float div)
{
	const int voxel = blockIdx.x * blockDim.x + threadIdx.x;
	if (voxel >= size)
		return;

	dataInOut[voxel] -= *value / div;
}
